#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

//block-thread 3D-3D
__global__ void testBlockThread9(int* c, const int* a, const int* b) {
    int threadId_3D = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int blockId_3D = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int i = threadId_3D + (blockDim.x * blockDim.y * blockDim.z) * blockId_3D;
    c[i] = b[i] - a[i];
}


void addWithCuda(int* c, const int* a, const int* b, unsigned int size) {
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    uint3 s1; s1.x = 5; s1.y = 2; s1.z = 2;
    uint3 s2; s2.x = size / 200; s2.y = 5; s2.z = 2;
    testBlockThread9 << <s1, s2 >> > (dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipGetLastError();
}


int main() {
    const int n = 1000;

    int* a = new int[n];
    int* b = new int[n];
    int* c = new int[n];
    int* cc = new int[n];

    for (int i = 0; i < n; i++) {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
        c[i] = b[i] - a[i];
    }

    addWithCuda(cc, a, b, n);

    FILE* fp = fopen("out.txt", "w");
    for (int i = 0; i < n; i++)
        fprintf(fp, "%d %d\n", c[i], cc[i]);
    fclose(fp);

    bool flag = true;
    for (int i = 0; i < n; i++) {
        if (c[i] != cc[i]) {
            flag = false;
            break;
        }
    }

    if (flag == false)
        printf("no pass");
    else
    {
        int sum = 0;
        sum=5/12;
        printf("sum=%d",sum);
        printf("pass");
    }


    hipDeviceReset();

    delete[] a;
    delete[] b;
    delete[] c;
    delete[] cc;

    getchar();
    return 0;
}
